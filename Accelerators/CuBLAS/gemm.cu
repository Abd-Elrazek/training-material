#include <functional>
#include <iostream>
#include <random>
#include <sys/time.h>

extern "C" {
#include "cl_params.h"
}

using namespace std;

#ifndef WITHOUT_GPU_XT
void compute_gpu_xt(int n, int nr_iters);
#endif
#ifndef WITHOUT_CPU
void compute_cpu(int n, int nr_iters);
#endif

int main(int argc, char *argv[]) {
    Params params;
    initCL(&params);
    parseCL(&params, &argc, &argv);
    dumpCL(stdout, "# ", &params);
#ifndef WITHOUT_GPU_XT
    if (!params.skip_gpu_xt)
        compute_gpu_xt(params.N, params.nr_iters);
#endif
#ifndef WITHOUT_CPU
    if (!params.skip_cpu)
        compute_cpu(params.N, params.nr_iters);
#endif
    finalizeCL(&params);
    return 0;
}

double compute_time(struct timeval start_time, struct timeval end_time) {
    return (end_time.tv_sec - start_time.tv_sec) +
                1.0e-6*(end_time.tv_usec - start_time.tv_usec);
}

double compute_sum(float *ptr, int size) {
    float sum {0.0};
    for (int i = 0; i < size; i++)
        sum += ptr[i];
    return sum;
}

#ifndef WITHOUT_GPU_XT
#include <cublasXt.h>
void compute_gpu_xt(int n, int nr_iters) {
    struct timeval start_time, end_time;
    float *a {new float[n*n]};
    float *b {new float[n*n]};
    float *c {new float[n*n]};
    if (!(a && b && c)) {
        cerr << "#error: can't allocate memory" << endl;
        exit(1);
    }
    for (int i = 0; i < n*n; i++) {
        a[i] = i + 1.0;
        b[i] = i + 10.0;
        c[i] = 0.0;
    }
    float alpha {1.0};
    float beta {1.0};
    cublasXtHandle_t handle;
    hipblasStatus_t status = cublasXtCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cerr << "# error: couldn't create handle: " << status << endl;
        exit(1);
    }
    const int nr_devices {2};
    int device_ids[] = {0, 1};
    cublasXtDeviceSelect(handle, nr_devices, device_ids);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cerr << "# error: couldn't select devices: " << status << endl;
        exit(1);
    }
    int block_dim;
    cublasXtGetBlockDim(handle, &block_dim);
    cout << "block dimension = " << block_dim << endl;
    gettimeofday(&start_time, NULL);
    for (int i = 0; i < nr_iters; i++) {
        status = cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                               n, n, n, &alpha, a, n, b, n, &beta, c, n);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            cerr << "# error: cublasXtSgemm returned: " << status << endl;
            exit(1);
        }
    }
    gettimeofday(&end_time, NULL);
    cout << "GPU_XT computation time = "
         << compute_time(start_time, end_time) << endl
         << "GPU_XT sum = " << compute_sum(c, n*n) << endl;
    delete a;
    delete b;
    delete c;
    cublasXtDestroy(handle);
}
#endif

#ifndef WITHOUT_CPU
#include <armadillo>
using namespace arma;
void compute_cpu(int n, int nr_iters) {
    struct timeval start_time, end_time;
    auto A = Mat<float>(n, n);
    auto B = Mat<float>(n, n);
    auto C = Mat<float>(n, n, fill::zeros);
    int k = 0;
    for (int j = 0; j < n; j++) {
        for (int i = 0; i < n; i++) {
            A(i, j) = k + 1.0;
            B(i, j) = k + 10.0;
            k++;
        }
    }
    gettimeofday(&start_time, NULL);
    for (int i = 0; i < nr_iters; i++) {
        C += A*B;
    }
    gettimeofday(&end_time, NULL);
    cout << "CPU computation time = "
         << compute_time(start_time, end_time) << endl
         << "CPU sum = " << compute_sum(C.memptr(), C.n_rows*C.n_cols)
         << endl;
}
#endif
