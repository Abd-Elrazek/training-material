
#include <hip/hip_runtime.h>
#include <functional>
#include <iostream>
#include <random>
#include <sys/time.h>
#include <cublasXt.h>

using namespace std;


double compute_time(struct timeval start_time, struct timeval end_time) {
    return (end_time.tv_sec - start_time.tv_sec) +
                1.0e-6*(end_time.tv_usec - start_time.tv_usec);
}

int main(int argc, char *argv[]) {
    const bool verbose = false;
    struct timeval start_time, end_time;
    int n = 5;
    if (argc > 1)
        n = atoi(argv[1]);
    float *a {new float[n*n]};
    float *b {new float[n*n]};
    float *c {new float[n*n]};
    if (!(a && b && c)) {
        cerr << "#error: can't allocate memory" << endl;
        return 1;
    }

    for (int i = 0; i < n*n; i++) {
        a[i] = i + 1.0;
        b[i] = i + 10.0;
        c[i] = 0.0;
    }
    float alpha {1.0};
    float beta {0.0};
    cublasXtHandle_t handle;
    hipblasStatus_t status = cublasXtCreate(&handle);
    const int nr_devices {2};
    int device_ids[] = {1, 1};
    cublasXtDeviceSelect(handle, nr_devices, device_ids);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cerr << "# error: couldn't create handle: " << status << endl;
        return 1;
    }
    gettimeofday(&start_time, NULL);
    status = cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                           n, n, n, &alpha, a, n, b, n, &beta, c, n);
    gettimeofday(&end_time, NULL);
    cout << "computation time = " << compute_time(start_time, end_time)
         << endl;
    float sum {0.0};
    for (int i = 0; i < n*n; i++)
        sum += c[i];
    cout << "sum = " << sum << endl;
    delete a;
    delete b;
    delete c;
    cublasXtDestroy(handle);
    return 0;
}
