#include <armadillo>
#include <functional>
#include <iostream>
#include <random>
#include <sys/time.h>
#include <cublasXt.h>

using namespace std;
using namespace arma;

double compute_time(struct timeval start_time, struct timeval end_time) {
    return (end_time.tv_sec - start_time.tv_sec) +
                1.0e-6*(end_time.tv_usec - start_time.tv_usec);
}

int main(int argc, char *argv[]) {
    float sum {0.0};
    struct timeval start_time, end_time;
    int n = 500;
    if (argc > 1)
        n = atoi(argv[1]);
    int nr_iters = 10;
    if (argc > 2)
        nr_iters = atoi(argv[2]);
    cout << n << "x" << n << " matrices, " << nr_iters << " iterations"
         << endl;
    float *a {new float[n*n]};
    float *b {new float[n*n]};
    float *c {new float[n*n]};
    if (!(a && b && c)) {
        cerr << "#error: can't allocate memory" << endl;
        return 1;
    }

    for (int i = 0; i < n*n; i++) {
        a[i] = i + 1.0;
        b[i] = i + 10.0;
        c[i] = 0.0;
    }
    float alpha {1.0};
    float beta {1.0};
    cout << "host matrices allocated and initialized" << endl;
    cublasXtHandle_t handle;
    hipblasStatus_t status = cublasXtCreate(&handle);
    const int nr_devices {2};
    int device_ids[] = {0, 1};
    cublasXtDeviceSelect(handle, nr_devices, device_ids);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cerr << "# error: couldn't create handle: " << status << endl;
        return 1;
    }
    gettimeofday(&start_time, NULL);
    for (int i = 0; i < nr_iters; i++) {
        status = cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                               n, n, n, &alpha, a, n, b, n, &beta, c, n);
    }
    gettimeofday(&end_time, NULL);
    cout << "GPU computation time = " << compute_time(start_time, end_time)
         << endl;
    sum = 0.0;
    for (int i = 0; i < n*n; i++)
        sum += c[i];
    cout << "GPU sum = " << sum << endl;
    delete a;
    delete b;
    delete c;
    cublasXtDestroy(handle);

    auto A = Mat<float>(n, n);
    auto B = Mat<float>(n, n);
    auto C = Mat<float>(n, n);
    int k = 0;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            A(i, j) = k + 1.0;
            B(i, j) = k + 10.0;
            C(i, j) = 0.0;
            k++;
        }
    }
    cout << "CPU matrices allocated and initialized" << endl;
    gettimeofday(&start_time, NULL);
    for (int i = 0; i < nr_iters; i++) {
        C += A*B;
    }
    gettimeofday(&end_time, NULL);
    cout << "CPU computation time = " << compute_time(start_time, end_time)
         << endl;
    sum = 0.0;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            sum += C(i, j);
        }
    }
    cout << "CPU sum = " << sum << endl;
    return 0;
}
